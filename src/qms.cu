#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <cstring>
#include <stdio.h>
#include <complex.h>
#include <cstdio>
//#include <bits/stdc++.h>
//#include <unistd.h>
#include <cmath>
#include <cassert>
#include <lapacke.h>
#include "Rand.hpp"
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include "io.hpp"
#include "parser.hpp"
#include "suqa.cuh"
#include "system.cuh"
#include "qms.cuh"

//XXX: test only, remove after
extern double *host_state_re, *host_state_im;
const double eev[8] = {-1.,-1.,-1.,-1.,-1.,-1.,3.,3.};


const int Amat_coo[16][2] = {{0,6},{0, 7},{1,6},{1,7},{2,4},{2, 5},{3,4},{3,5},{4,2},{4, 3},{5,2},{5,3},{6,0},{6, 1},{7,0},{7,1}};
const double Amat_val[16][2] = {{1,0},{0,-1},{0,1},{1,0},{1,0},{0,-1},{0,1},{1,0},{1,0},{0,-1},{0,1},{1,0},{1,0},{0,-1},{0,1},{1,0}};

using namespace std;





#define NUM_THREADS 128
#define MAXBLOCKS 65535
uint suqa::threads;
uint suqa::blocks;
hipStream_t suqa::stream1, suqa::stream2;


string tag;
// simulation parameters
double beta_therm;
double h;
int thermalization;

// defined in src/system.cu
void init_state(ComplexVec& state, uint Dim);

arg_list args;

void save_measures(string outfilename){
    FILE * fil = fopen(outfilename.c_str(), "a");
    for(uint ei = 0; ei < qms::E_measures.size(); ++ei){
        fprintf(fil, "%.16lg %.16lg\n", qms::E_measures[ei], qms::X_measures[ei]);
    }
    fclose(fil);
    qms::E_measures.clear();
    qms::X_measures.clear();
}

void deallocate_state(ComplexVec& state){
    if(state.data!=nullptr){
        HANDLE_CUDACALL(hipFree(state.data));
    }
    state.vecsize=0U;
}

void allocate_state(ComplexVec& state, uint Dim){
    if(state.data!=nullptr or Dim!=state.vecsize)
        deallocate_state(state);


    state.vecsize = Dim; 
    HANDLE_CUDACALL(hipMalloc((void**)&(state.data), 2*state.vecsize*sizeof(double)));
    // allocate both using re as offset, and im as access pointer.
    state.data_re = state.data;
    state.data_im = state.data_re + state.vecsize;
}

void save_rho(string rhomat_fname,double rho_proj[8][8][2] , int iiii){
    cout<<"saving rho"<<endl;
    FILE * outrho = fopen(rhomat_fname.c_str(),"w");

    fprintf(outrho,"%d\n",iiii);
    for(int i=0;i<8;++i){
	for(int j=0;j<8;++j) for(int k=0;k<2;++k){
	    fprintf(outrho,"%.12lg ",rho_proj[i][j][k]);
	}
	fprintf(outrho,"\n");
    }
    fclose(outrho);
}


int main(int argc, char** argv){
    if(argc < 8){
        printf("usage: %s <beta> <g_beta> <metro steps> <reset each> <num state qbits> <num ene qbits> <output file path> [--max-reverse <max reverse attempts> (20)] [--seed <seed> (random)] [--ene-min <min energy> (0.0)] [--ene-max <max energy> (1.0)] [--PE-steps <steps of PE evolution> (10)] [--thermalization <steps> (100)] [--record-reverse] [--walltime (0)]\n", argv[0]);
        exit(1);
    }

    parse_arguments(args, argc, argv);

    tag = args.tag;
    beta_therm = args.beta;
    g_beta = args.g_beta; // defined as extern in system.cuh
    thermalization = args.thermalization;
    qms::metro_steps = (uint)args.metro_steps;
    qms::reset_each = (uint)args.reset_each;
    qms::state_qbits = (uint)args.state_qbits;
    qms::ene_qbits = (uint)args.ene_qbits;
    string outfilename(args.outfile);
    qms::max_reverse_attempts = (uint)args.max_reverse_attempts;
    qms::n_phase_estimation = args.pe_steps;
    qms::record_reverse= args.record_reverse;
    qms::iseed = args.seed;
    if(qms::iseed>0)
        qms::rangen.set_seed(qms::iseed);
    
    qms::iseed = qms::rangen.get_seed();

    qms::nqubits = qms::state_qbits + qms::ene_qbits + 1;
    qms::Dim = (1U << qms::nqubits);
    qms::ene_levels = (1U << qms::ene_qbits);
    qms::state_levels = (1U << qms::state_qbits);

    qms::t_PE_shift = args.ene_min;
    qms::t_PE_factor = (qms::ene_levels-1)/(double)(qms::ene_levels*(args.ene_max-args.ene_min)); 
    qms::t_phase_estimation = qms::t_PE_factor*8.*atan(1.0); // 2*pi*t_PE_factor

    suqa::threads = NUM_THREADS;
    suqa::blocks = (qms::Dim+suqa::threads-1)/suqa::threads;
    if(suqa::blocks>MAXBLOCKS) suqa::blocks=MAXBLOCKS;

    
    // Banner
    suqa::print_banner();
    cout<<"arguments:\n"<<args<<endl;

    auto t_start = std::chrono::high_resolution_clock::now();

    // Initialization of utilities
    suqa::setup(qms::Dim);
    qms::setup(beta_therm);

    // Initialization:
    // known eigenstate of the system (see src/system.cu)
    
    allocate_state(qms::gState, qms::Dim);
    init_state(qms::gState,qms::Dim);


    //TODO: make it an args option?
    uint perc_mstep = (qms::metro_steps+19)/20; // batched saves
    
    uint count_accepted = 0U;
//    if(!file_exists(outfilename.c_str())){
//        FILE * fil = fopen(outfilename.c_str(), "w");
//        fprintf(fil, "# E A\n");
//        fclose(fil);
//    }

    //XXX: systematic test
    int iiii=0;
    double rho_proj[8][8][2];
    
    string rhomat_fname="rho_mat_qms_"+tag+".txt"; //#_b"+to_string(beta_therm)+"_rt_"+to_string(qms::reset_each)+".txt";

    if( access( rhomat_fname.c_str(), F_OK ) != -1 ){
        printf("%s exists\n",rhomat_fname.c_str());
        FILE * fil = fopen(rhomat_fname.c_str(), "r");
        if(fscanf(fil,"%d\n",&iiii)!=1){ printf("Wrong reading!\n"); exit(1);}
        printf("Restarting from idx: %d\n",iiii);
        for(int i=0;i<8;++i){
            for(int j=0;j<8;++j) for(int k=0;k<2;++k){
                if(fscanf(fil,"%lg ",&rho_proj[i][j][k])!=1){ printf("Wrong reading!\n"); exit(1);}
            }
            if(fscanf(fil,"\n") !=0){ printf("Wrong reading!\n"); exit(1);}
        }
        fclose(fil);
        printf("Loading previous rho matrix");
    }else{
        for(uint i=0; i<8; ++i)  for(uint j=0; j<8; ++j) for(uint k=0; k<2; ++k) rho_proj[i][j][k]=0.0;
    }

    double TrDist_discrepancy, TrDist_discrepancy_prev=100000.;
    double Energy_discrepancy, Energy_discrepancy_prev=100000.;
    double Aoper_discrepancy, Aoper_discrepancy_prev=100000.;

    int TrDist_ctr=0;
    bool TrDist_first10done = false;
    std::vector<double> TrDist_recents(10);

    // partition function precomputation
    double Z=0.0;
    double E_sng_exact=0.0,E_sqr_exact;
    for(uint i=0; i<8; ++i){
        Z+=exp(-beta_therm*eev[i]);
    }
    for(uint i=0;i<8;++i){
//        rho_diff_re[cci]=rho_proj[i][j][0]*rho_proj[i][j][0]/(sampling*sampling);
        E_sng_exact+=eev[i]*exp(-beta_therm*eev[i])/Z;
        E_sqr_exact+=eev[i]*eev[i]*exp(-beta_therm*eev[i])/Z;
    }

    auto t_prev = std::chrono::high_resolution_clock::now();

    bool take_measure;
    uint s0 = 0U;
    for(uint s = 0U; s < qms::metro_steps; ++s){
        DEBUG_CALL(cout<<"metro step: "<<s<<endl);
        take_measure = (s>s0+(uint)thermalization and (s-s0)%qms::reset_each ==0U);

        double tmp_rho[8][8][2];
        for(uint i=0; i<8; ++i)  for(uint j=0; j<8; ++j) for(uint k=0; k<2; ++k) tmp_rho[i][j][k]=0.0;

        int ret = qms::metro_step(take_measure, tmp_rho);

        // check conditions of measurement
        if(ret<0){ // failed rethermalization, reinitialize state
            init_state(qms::gState, qms::Dim);
            //ensure new rethermalization
            s0 = s+1; 
        }

        if(take_measure and (ret==2 or ret==4)){
            // measure rho as weighted average of eigenstates projectors
//            DUMP_STATE(qms::gState);

            iiii++;

//            printf("rho:\n");
            for(uint i=0;i<8;++i){
                for(uint j=0;j<8;++j){
                    rho_proj[i][j][0]+=tmp_rho[i][j][0];
                    rho_proj[i][j][1]+=tmp_rho[i][j][1];
//                    printf("(%.2lg %.2lg) ",rho_proj[i][j][0]/iiii,rho_proj[i][j][1]/iiii); 
                }
//                printf("\n");
            }

            auto t_tmp = std::chrono::high_resolution_clock::now();
            double secs_aft = (1./1000.)*std::chrono::duration<double, std::milli>(t_tmp-t_prev).count();
//            if(iiii%100==0){
            if(secs_aft>2.0){
                t_prev=t_tmp;
                lapack_complex_double rho_diff[64]; //,rho_A_prod[8][8];
                double E_sng=0.0, E_sqr=0.0;
                double E_isolated=0.0;
                uint cci=0;
                for(uint i=0;i<8;++i) for(uint j=0;j<8;++j){
                //        rho_diff_re[cci]=rho_proj[i][j][0]*rho_proj[i][j][0]/(sampling*sampling);
                    rho_diff[cci]=lapack_make_complex_double(rho_proj[i][j][0]/iiii,rho_proj[i][j][1]/iiii);
                    //rho_A_prod[i][j]=0.0;

                    if(i==j){
                        E_isolated+=eev[i]*tmp_rho[i][j][0];
                        E_sng+=eev[i]*rho_proj[i][j][0]/iiii;
                        E_sqr+=eev[i]*eev[i]*rho_proj[i][j][0]/iiii;
                        
                        rho_diff[cci]-=exp(-beta_therm*eev[i])/Z;
                    }
                //        printf("%.4lg %.4lg %.4lg\n",rho_proj[i][j][0]*rho_proj[i][j][0]/sampling,exp(-qsa::beta*eev[i])/Z, rho_diff_re[cci]);
                    cci++;
                }

//                printf("E_iso: %.6lg, E_meas: %.6lg\n",E_isolated,qms::E_measures.back());

                double A_sng=0.0, A_sqr=0.0;
                double A_sng_exact=0.0;
                for(int coeff=0;coeff<16;++coeff){
                    int i = Amat_coo[coeff][0];        
                    int k = Amat_coo[coeff][1];        
                    double a_ik= Amat_val[coeff][0]; // A_ik

                    for(int n=0; n<8; ++n){
                        for(int m=0; m<8; ++m){
                            A_sng+=ees[n][k]*rho_proj[n][m][0]/iiii*ees[m][i]*a_ik;
//                            A_sqr+=ees[n][k]*rho_proj[n][m][0]/iiii*ees[m][i]*a_ik*a_ik;
                        }
                        A_sng_exact+=ees[n][k]*exp(-beta_therm*eev[n])/Z*ees[n][i]*a_ik;
                    }

                    for(int coeff2=0;coeff2<16;++coeff2){
                        int p = Amat_coo[coeff2][0];        
                        int l = Amat_coo[coeff2][1];        
                        if(p!=k) continue;
                        double a_kl= Amat_val[coeff2][0]; // A_kl

                        for(int n=0; n<8; ++n) for(int m=0; m<8; ++m){
                                A_sqr+=ees[n][l]*rho_proj[n][m][0]/iiii*ees[m][i]*a_ik*a_kl;
                        }
                    }
                }

                printf("A_sng=%.8lg\tA_sqr=%.8lg; A_sng_exact=%.8lg\n",A_sng,A_sqr,A_sng_exact);
                double A_std = sqrt((A_sqr-A_sng*A_sng)/iiii);
                Aoper_discrepancy = (A_sng-A_sng_exact)/A_std;


                double E_std = sqrt((E_sqr-E_sng*E_sng)/iiii);
                double E_std_exact = sqrt((E_sqr_exact-E_sng_exact*E_sng_exact)/iiii);
                printf("E_exact: %.12lg, E: %.12lg, dE: %.12lg; rel_discr: %.12lg\n",E_sng_exact,E_sng,E_std_exact,abs(E_sng-E_sng_exact)/E_std_exact);

                Energy_discrepancy = (E_sng-E_sng_exact)/E_std;

                // eigensolver
                lapack_int n=8;
//                lapack_int lwork=2*n-1;
//                double work[lwork];
//                double rwork[3*n-2];
                char jobz = 'N', uplo = 'U';
                lapack_int info;
                double dist_eigs[n];
//                LAPACK_dsyev("N", "U", &n, rho_diff_re, &n, dist_eigs,work,&lwork,&info);
                info = LAPACKE_zheev(LAPACK_COL_MAJOR,jobz,uplo, n, rho_diff, n, dist_eigs);

                if(info){
                    printf("Error in eigenvalue routine\n");
                }else{
                    double tr_dist=0.0;
//                    printf("dist eigs:\n");
                    for(uint i=0; i<8; ++i){
//                        printf("%.10lg\n",dist_eigs[i]);
                        tr_dist += abs(dist_eigs[i]);
                    }
                    tr_dist*=0.5;

//                    printf("Trace distance: %.12lg\n",tr_dist);
                    TrDist_discrepancy = tr_dist;
                }

                TrDist_recents[TrDist_ctr]=TrDist_discrepancy;
                TrDist_ctr=(TrDist_ctr+1)%10;
                if(!TrDist_first10done and TrDist_ctr==0) TrDist_first10done = true;

                double TrDist_ave=0.0, TrDist_fluct=0.0;
                if(TrDist_first10done){
                    for(int trdi=0;trdi<10;++trdi){
                        TrDist_ave+=TrDist_recents[trdi];
                        TrDist_fluct+=TrDist_recents[trdi]*TrDist_recents[trdi];
                    }
                    TrDist_ave/=10.0;
                    TrDist_fluct-=10.0*TrDist_ave*TrDist_ave;
                    TrDist_fluct*=10./9.;
                    TrDist_fluct=sqrt(TrDist_fluct);
                }


                double thr_discr=3.0;

//                printf("%.8lg+-%.8lg (%.8lg);\t%.8lg+-%.8lg (%.8lg)|%.8lg\t%.8lg\t%.8lg\t%.8lg\n",E_sng,E_std,E_sng_exact,A_sng,A_std,A_sng_exact,TrDist_ave,TrDist_fluct,Energy_discrepancy,Aoper_discrepancy);
//                printf("%.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg\n",E_sng,E_std,E_sng_exact,A_sng,A_std,A_sng_exact,TrDist_ave,TrDist_fluct,Energy_discrepancy,Aoper_discrepancy);

                { // best res file
                    string bestres_outfilename="best_systest_"+outfilename+"_b"+to_string(beta_therm)+"_gb"+to_string(g_beta)+"_em"+to_string(args.ene_min)+"_eM"+to_string(args.ene_max)+"_nqe"+to_string(qms::ene_qbits)+"_rt"+to_string(qms::reset_each);
//                    if( access( outfilename.c_str(), F_OK ) == -1 ){
//                        FILE * fil = fopen(outfilename.c_str(), "w");
//                        //        fprintf(fil, "# E%s\n",(Xmatstem!="")?" A":"");
//                        fprintf(fil, "#beta retherm E_mean E_err E_exact A_mean A_err A_exact TrDist TrDist_err\n");
//                        fclose(fil);
//                    }
                    FILE * fil = fopen(bestres_outfilename.c_str(), "w");
                    printf("%d %.8lg %.8lg %.8lg %.8lg %d %d %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg\n",iiii, beta_therm, g_beta, args.ene_min, args.ene_max, qms::ene_qbits, qms::reset_each, E_sng,E_std,E_sng_exact,A_sng,A_std,A_sng_exact,TrDist_ave,TrDist_fluct,Energy_discrepancy,Aoper_discrepancy);
                    fprintf(fil,"%s %s %s %s %s %s %s %s %s %s %s %s %s %s %s %s %s\n","iiii", "beta_therm", "g_beta", "args.ene_min", "args.ene_max", "qms::ene_qbits", "qms::reset_each", "E_sng", "E_std", "E_sng_exact", "A_sng", "A_std", "A_sng_exact", "TrDist_ave", "TrDist_fluct", "Energy_discrepancy", "Aoper_discrepancy");
                    fprintf(fil,"%d %.8lg %.8lg %.8lg %.8lg %d %d %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg %.8lg\n",iiii, beta_therm, g_beta, args.ene_min, args.ene_max, qms::ene_qbits, qms::reset_each, E_sng,E_std,E_sng_exact,A_sng,A_std,A_sng_exact,TrDist_ave,TrDist_fluct,Energy_discrepancy,Aoper_discrepancy);
                    fclose(fil);
                }

                if(    (abs(1.0-TrDist_discrepancy_prev/TrDist_discrepancy)<1e-4)
                   and (abs(Energy_discrepancy)>thr_discr or abs(1.0-Energy_discrepancy_prev/Energy_discrepancy)<1e-6)
                   and (abs(Aoper_discrepancy)>thr_discr or abs(1.0-Aoper_discrepancy_prev/Aoper_discrepancy)<1e-6)){
                    printf("Discrepancies converged\n");

                    if( access( outfilename.c_str(), F_OK ) == -1 ){
                        FILE * fil = fopen(outfilename.c_str(), "w");
                        //        fprintf(fil, "# E%s\n",(Xmatstem!="")?" A":"");
                        fprintf(fil, "#beta retherm E_mean E_err E_exact A_mean A_err A_exact TrDist TrDist_err\n");
                        fclose(fil);
                    }
                    FILE * fil = fopen(outfilename.c_str(), "a");
                    fprintf(fil, "%.8lg\t%2d\t%.12lg\t%.12lg\t%.12lg\t%.12lg\t%.12lg\t%.12lg\t%.12lg\t%.12lg\n", 
                            beta_therm, qms::reset_each, E_sng, E_std, E_sng_exact, A_sng, A_std, A_sng_exact, TrDist_ave, TrDist_fluct);
                    fclose(fil);

		    save_rho(rhomat_fname,rho_proj,iiii);
                    break;
                }

                TrDist_discrepancy_prev = TrDist_discrepancy;
                Energy_discrepancy_prev = Energy_discrepancy;
                Aoper_discrepancy_prev = Aoper_discrepancy;

            }

        }

        if(ret==1 or ret==2){
            count_accepted++;
        }
        if(s%perc_mstep==0){
            cout<<"iteration: "<<s<<"/"<<qms::metro_steps<<endl;
            save_measures((outfilename+"_meas").c_str());
        }

        auto t_mid = std::chrono::high_resolution_clock::now();
        double secs_passed = (1./1000.)*std::chrono::duration<double, std::milli>(t_mid-t_start).count();
        if((args.walltime>0 and secs_passed>args.walltime) or access( "stop", F_OK ) != -1 ){
            remove("stop");
            // save rho and stop
            
	    save_rho(rhomat_fname,rho_proj,iiii);

            printf("Closing due to walltime limit reached\n");
            break;
        }
    }

    cout<<endl;
    printf("\n\tacceptance: %3.2lg%%\n",(count_accepted/static_cast<double>(qms::metro_steps))*100.0);


    deallocate_state(qms::gState);
    qms::clear();
    suqa::clear();

    cout<<"\nall fine :)\n"<<endl;



    if(qms::record_reverse){
        FILE * fil_rev = fopen((outfilename+"_revcounts").c_str(), "w");

        for(uint i = 0; i < qms::reverse_counters.size(); ++i){
            fprintf(fil_rev, "%d %d\n", i, static_cast<int>(qms::reverse_counters[i]));
        }
        fclose(fil_rev);
    }

    cout<<"\n\tSuqa!\n"<<endl;

    auto t_end = std::chrono::high_resolution_clock::now();
    double secs_passed = (1./1000.)*std::chrono::duration<double, std::milli>(t_end-t_start).count();
	cout<<"All [DONE] in "<<secs_passed<<" seconds"<<endl;

    return 0;
}
